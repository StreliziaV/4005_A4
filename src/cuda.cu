#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <math.h>
#include <stdio.h>
#include <stdlib.h>
// #include <chrono>

#ifdef GUI
#include <GL/glut.h>
#include <GL/gl.h>
#include <GL/glu.h>
#endif

#include "./headers/physics.h"


int block_size = 512; // cuda thread block size
int size; // problem size
__device__ int dsize = 10;


__global__ void initialize(float *data) {
    // TODO: intialize the temperature distribution (in parallelized way)
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= dsize * dsize) return;
    data[idx] = wall_temp;
}


__global__ void generate_fire_area(bool *fire_area){
    // TODO: generate the fire area (in parallelized way)
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= dsize * dsize) return;
    int i = idx / dsize;
    int j = idx % dsize;
    fire_area[idx] = 0;

    float fire1_r2 = fire_size * fire_size;
    int a = i - dsize / 2;
    int b = j - dsize / 2;
    int r2 = 0.5 * a * a + 0.8 * b * b - 0.5 * a * b;
    if (r2 < fire1_r2) fire_area[i * dsize + j] = 1;

    float fire2_r2 = (fire_size / 2) * (fire_size / 2);
    a = i - 1 * dsize / 3;
    b = j - 1 * dsize / 3;
    r2 = a * a + b * b;
    if (r2 < fire2_r2) fire_area[i * dsize + j] = 1;
}


__global__ void update(float *data, float *new_data) {
    // TODO: update temperature for each point  (in parallelized way)
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= dsize * dsize) return;
    int i = idx / dsize;
    int j = idx % dsize;
    if (i == 0 || i == dsize - 1 || j == 0 || j == dsize - 1) return;

    float up = data[idx - dsize];
    float down = data[idx + dsize];
    float left = data[idx - 1];
    float right = data[idx + 1];
    float new_val = (up + down + left + right) / 4;
    new_data[idx] = new_val;
}


__global__ void maintain_wall(float *data) {
    // TODO: maintain the temperature of the wall (sequential is enough)
    data[0] = dsize;
}


__global__ void maintain_fire(float *data, bool *fire_area) {
    // TODO: maintain the temperature of the fire (in parallelized way)
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= dsize * dsize) return;
    
    if (fire_area[idx]) data[idx] = fire_temp;
}

__global__ void ini_size(int *data) {
    dsize = data[0];
}

#ifdef GUI
__global__ void data2pixels(float *data, GLubyte* pixels){
    // TODO: convert rawdata (large, size^2) to pixels (small, resolution^2) for faster rendering speed (in parallelized way)
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx >= resolution * resolution) return;
    float factor_data_pixel = (float) dsize / resolution;
    float factor_temp_color = (float) 255 / fire_temp;
    int x = idx / resolution;
    int y = idx % resolution;

    int idx_pixel = idx * 3;
    int x_raw = x * factor_data_pixel;
    int y_raw = y * factor_data_pixel;
    int idx_raw = x_raw * dsize + y_raw;
    float temp = data[idx_raw];
    int color =  ((int) temp / 5 * 5) * factor_temp_color;
    pixels[idx_pixel] = color;
    pixels[idx_pixel + 1] = 255 - color;
    pixels[idx_pixel + 2] = 255 - color;
}


void plot(GLubyte* pixels){
    // visualize temprature distribution
    #ifdef GUI
    glClear(GL_COLOR_BUFFER_BIT);
    glDrawPixels(resolution, resolution, GL_RGB, GL_UNSIGNED_BYTE, pixels);
    glutSwapBuffers();
    #endif
}
#endif


void master() {
    float *data_odd;
    float *data_even;
    bool *fire_area;

    hipMalloc(&data_odd, size * size * sizeof(float));
    hipMalloc(&data_even, size * size * sizeof(float));
    hipMalloc(&fire_area, size * size * sizeof(bool));

    #ifdef GUI
    GLubyte *pixels;
    GLubyte *host_pixels;
    host_pixels = new GLubyte[resolution * resolution * 3];
    hipMalloc(&pixels, resolution * resolution * 3 * sizeof(GLubyte));
    #endif

    int n_block_size = size * size / block_size + 1;
    int n_block_resolution = resolution * resolution / block_size + 1;

    initialize<<<n_block_size, block_size>>>(data_odd);
    generate_fire_area<<<n_block_size, block_size>>>(fire_area);
    maintain_fire<<<n_block_size, block_size>>>(data_odd, fire_area);
    
    int count = 1;
    double total_time = 0;

    while (true){
        // std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();

        // TODO: modify the following lines to fit your need.
        if (count % 2 == 1) {
            update<<<n_block_size, block_size>>>(data_odd, data_even);
            maintain_fire<<<n_block_size, block_size>>>(data_even, fire_area);
            maintain_wall<<<1, 1>>>(data_even);
        } else {
            update<<<n_block_size, block_size>>>(data_even, data_odd);
            maintain_fire<<<n_block_size, block_size>>>(data_odd, fire_area);
            maintain_wall<<<1, 1>>>(data_odd);
        }

        // std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();
        // double this_time = std::chrono::duration<double>(t2 - t1).count();
        // total_time += this_time;
        // printf("Iteration %d, elapsed time: %.6f\n", count, this_time);

        count++;
        if (count > 1000) break;

        #ifdef GUI
        if (count % 2 == 1) {
            data2pixels<<<n_block_resolution, block_size>>>(data_even, pixels);
        } else {
            data2pixels<<<n_block_resolution, block_size>>>(data_odd, pixels);
        }
        hipMemcpy(host_pixels, pixels, resolution * resolution * 3 * sizeof(GLubyte), hipMemcpyDeviceToHost);
        plot(host_pixels);
        #endif

    }

    printf("Converge after %d iterations, elapsed time: %.6f, average computation time: %.6f\n", count-1, total_time, (double) total_time / (count-1));


    hipFree(data_odd);
    hipFree(data_even);
    hipFree(fire_area);

    #ifdef GUI
    hipFree(pixels);
    delete[] host_pixels;
    #endif
    
}


int main(int argc, char *argv[]){
    
    size = atoi(argv[1]);
    int* temp_d;
    int* temp_h = &size;
    hipMalloc(&temp_d, sizeof(int));
    hipMemcpy(temp_d, temp_h, sizeof(int), hipMemcpyHostToDevice);
    ini_size<<<1, 1>>>(temp_d);
    #ifdef GUI
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_RGB | GLUT_SINGLE);
    glutInitWindowPosition(0, 0);
    glutInitWindowSize(resolution, resolution);
    glutCreateWindow("Heat Distribution Simulation CUDA Implementation");
    gluOrtho2D(0, resolution, 0, resolution);
    #endif

    master();

    // float* d;
    // float* d_host = new float[5];
    // hipMalloc(&d, 5 * sizeof(float));
    // maintain_wall<<<1, 1>>>(d);
    // hipMemcpy(d_host, d, 5 * sizeof(float), hipMemcpyDeviceToHost);
    // printf("%f \n", d_host[0]);

    printf("Student ID: 119010369\n"); // replace it with your student id
    printf("Name: Bodong Yan\n"); // replace it with your name
    printf("Assignment 4: Heat Distribution CUDA Implementation\n");

    return 0;

}


